#include "hip/hip_runtime.h"
#include "example.h"
#include "hip/hip_runtime.h"



namespace torch2trt_plugins {


template<typename T>
__global__ void exampleKernel(T *x, int size) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < size) {
        x[index] *= 2;
    }
}

template __global__ void exampleKernel<float>(float *, int);
template __global__ void exampleKernel<int>(int *, int);


template<typename T>
void exampleFuncton(T *x, int size) {
    int nThreads = 32;
    int nBlocks = (size / 32) + 1;
    exampleKernel<<<nBlocks, nThreads>>>(x, size);
}

template void exampleFuncton<float>(float *, int);
template void exampleFuncton<int>(int *, int);


ExamplePlugin::ExamplePlugin() {

}

ExamplePlugin::~ExamplePlugin() {

}

AsciiChar const * ExamplePlugin::getPluginType() const noexcept {
    return "ExamplePlugin";
}

AsciiChar const * ExamplePlugin::getPluginVersion() const noexcept {
    return "1";
}

int32_t ExamplePlugin::getNbOutputs() const noexcept {
    return 1;
}

Dims ExamplePlugin::getOutputDimensions(int32_t index, Dims const* inputs, int32_t nbInputDims) noexcept {
    return inputs[0];
}

bool ExamplePlugin::supportsFormat(DataType type, PluginFormat format) const noexcept {
    return (type == DataType::kFLOAT) || (type == DataType::kINT32);
}

void ExamplePlugin::configureWithFormat(Dims const* inputDims, int32_t nbInputs, Dims const* outputDims, int32_t nbOutputs,
    DataType type, PluginFormat format, int32_t maxBatchSize) noexcept {
    Dims d = inputDims[0];
    this->inputSize = 1;
    for (int i = 0; i < d.nbDims; i++) {
        this->inputSize *= d.d[i];
    }
};

int32_t ExamplePlugin::initialize() noexcept {
    return 0;
};

void ExamplePlugin::terminate() noexcept {

};

size_t ExamplePlugin::getWorkspaceSize(int32_t maxBatchSize) const noexcept {
    return 0;
};

int32_t ExamplePlugin::enqueue(int32_t batchSize, void const* const* inputs, void* const* outputs, void* workspace,
    hipStream_t stream) noexcept {
    return 0;
};

size_t ExamplePlugin::getSerializationSize() const noexcept {
    return 0;
};

void ExamplePlugin::serialize(void* buffer) const noexcept {

};

void ExamplePlugin::destroy() noexcept {

};

IPluginV2* ExamplePlugin::clone() const noexcept { 
    return nullptr; 
};

void ExamplePlugin::setPluginNamespace(AsciiChar const* pluginNamespace) noexcept {

};

AsciiChar const* ExamplePlugin::getPluginNamespace() const noexcept {
    return "";
};

}