#include "hip/hip_runtime.h"
#include "reflection_pad_2d_plugin.h"


template<typename T>
__global__ void cuda_double_kernel(T *x) {
    *x = (*x) * 2;
}

template __global__ void cuda_double_kernel<float>(float *);


template<typename T>
void cuda_double(T *x) {
    cuda_double_kernel<<<1, 1>>>(x);
}

template void cuda_double<float>(float *);