#include <catch2/catch_all.hpp>
#include "hip/hip_runtime.h"
#include "example.h"


TEMPLATE_TEST_CASE("Example cuda test", "[example][template]" , int) {
    TestType x_cpu;
    TestType *x_gpu;
    
    hipMalloc(&x_gpu, sizeof(int));
    x_cpu = 2;
    hipMemcpy(x_gpu, &x_cpu, sizeof(int), hipMemcpyHostToDevice);
    cuda_double<TestType>(x_gpu);
    hipMemcpy(&x_cpu, x_gpu, sizeof(int), hipMemcpyDeviceToHost);
    REQUIRE(x_cpu == 4);
}